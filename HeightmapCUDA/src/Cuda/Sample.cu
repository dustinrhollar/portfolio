#include "hip/hip_runtime.h"

// All it does is set the framebuffer color to a specific value
GLOBAL
void SimpleKernel(vec3 *fb, i32 max_x, i32 max_y)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;
    int pixel_index = j * max_x + i;
    
    if ((i >= max_x) || (j >= max_y)) return;
    
    fb[pixel_index] = vec3(1,0,0);
}

// pbo: OpenGl texture image
void SimpleEntry(uchar4 *pbo, i32 texture_width, i32 texture_height)
{
    int nx = texture_width;
    int ny = texture_height;
    int tx = 8;
    int ty = 8;
    int num_pixels = nx * ny;
    size_t fb_size = sizeof(vec3) * num_pixels;
    dim3 blocks(nx/tx+1, ny/ty+1);
    dim3 threads(tx,ty);
    
    // Image to write to
    vec3* cudaimage = NULL;
    CheckCudaErrors(hipMalloc((void**)&cudaimage, fb_size));
    
    SimpleKernel<<<blocks, threads>>>(cudaimage, nx, ny);
    CheckCudaErrors(hipGetLastError());
    CheckCudaErrors(hipDeviceSynchronize());
    
    SendImageToPboRgb<<<blocks, threads>>>(pbo, nx, ny, cudaimage);
    CheckCudaErrors(hipGetLastError());
    CheckCudaErrors(hipDeviceSynchronize());
    
    CheckCudaErrors(hipFree(cudaimage));
    hipDeviceReset();
}
