#include "hip/hip_runtime.h"

// Proof for range of Perlin Function can be found here:
//https://digitalfreepen.com/2017/06/20/range-perlin-noise.html
//
// sqrt(N / 4)
// where N is the number of dimensions
#define PERLIN_3D_RANGE 0.86602540378f

// Hashing function (used for fast on-device pseudorandom numbers for randomness in noise)
DEVICE FORCE_INLINE
u32 Hash(unsigned int seed)
{
    seed = (seed + 0x7ed55d16) + (seed << 12);
    seed = (seed ^ 0xc761c23c) ^ (seed >> 19);
    seed = (seed + 0x165667b1) + (seed << 5);
    seed = (seed + 0xd3a2646c) ^ (seed << 9);
    seed = (seed + 0xfd7046c5) + (seed << 3);
    seed = (seed ^ 0xb55a4f09) ^ (seed >> 16);
    return seed;
}

// Random unsigned int for a grid coordinate [0, MAXUINT]
DEVICE FORCE_INLINE
u32 RandomIntGrid(i32 x, i32 y, i32 z, i32 seed = 0)
{
    return Hash((unsigned int)(x * 1723.0f + y * 93241.0f + z * 149812.0f + 3824 + seed));
}

DEVICE FORCE_INLINE
r32 Fade(r32 t) 
{
    return t * t * t * (t * (t * 6.0f - 15.0f) + 10.0f); 
}

DEVICE FORCE_INLINE
r32 Grad(int hash, r32 x, r32 y, r32 z) {
    switch (hash & 0xF)
    {
		case 0x0: return x + y;
		case 0x1: return -x + y;
		case 0x2: return x - y;
		case 0x3: return -x - y;
		case 0x4: return x + z;
		case 0x5: return -x + z;
		case 0x6: return x - z;
		case 0x7: return -x - z;
		case 0x8: return y + z;
		case 0x9: return -y + z;
		case 0xA: return y - z;
		case 0xB: return -y - z;
		case 0xC: return y + x;
		case 0xD: return -y + z;
		case 0xE: return y - x;
		case 0xF: return -y - z;
		default: return 0; // never happens
    }
}

// Linearly interpolate between two float values
__device__ float Lerp(float a, float b, float ratio)
{
    return a * (1.0f - ratio) + b * ratio;
}

DEVICE
r32 SamplePerlinNoise(r32 x, r32 y, r32 z, r32 scale, i32 seed)
{
    r32 fseed = (r32)seed;
    
    // TODO(Dustin): Scale
    x *= scale;
    y *= scale;
    z *= scale;
    
    r32 ix = floorf(x);
    r32 iy = floorf(y);
    r32 iz = floorf(z);
    
    x -= ix;                                // FIND RELATIVE X,Y,Z
    y -= iy;                                // OF POINT IN CUBE.
    z -= iz;
    
    r32 u = Fade(x); // compute fade curves
    r32 v = Fade(y);
    r32 w = Fade(z);
    
    // Get the influence values from each corner of the cube
    r32 i000 = Grad(RandomIntGrid(ix, iy, iz, fseed), x, y, z);
    r32 i100 = Grad(RandomIntGrid(ix + 1.0f, iy, iz, fseed), x - 1.0f, y, z);
    r32 i010 = Grad(RandomIntGrid(ix, iy + 1.0f, iz, fseed), x, y - 1.0f, z);
    r32 i110 = Grad(RandomIntGrid(ix + 1.0f, iy + 1.0f, iz, fseed), x - 1.0f, y - 1.0f, z);
    r32 i001 = Grad(RandomIntGrid(ix, iy, iz + 1.0f, fseed), x, y, z - 1.0f);
    r32 i101 = Grad(RandomIntGrid(ix + 1.0f, iy, iz + 1.0f, fseed), x - 1.0f, y, z - 1.0f);
    r32 i011 = Grad(RandomIntGrid(ix, iy + 1.0f, iz + 1.0f, fseed), x, y - 1.0f, z - 1.0f);
    r32 i111 = Grad(RandomIntGrid(ix + 1.0f, iy + 1.0f, iz + 1.0f, fseed), x - 1.0f, y - 1.0f, z - 1.0f);
    
    // Interpolate
    float x00 = Lerp(i000, i100, u);
    float x10 = Lerp(i010, i110, u);
    float x01 = Lerp(i001, i101, u);
    float x11 = Lerp(i011, i111, u);
    
    float y0 = Lerp(x00, x10, v);
    float y1 = Lerp(x01, x11, v);
    
    float avg = Lerp(y0, y1, w);
    
    return avg;
}

GLOBAL
void PerlinKernel(r32 *fb, float3 dims, i32 seed, i32 n, r32 scale, r32 lacunarity, r32 decay)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;
    int pixel_index = j * dims.x + i;
    
    float3 pos = make_float3((r32)i / (r32)dims.x, (r32)j / (r32)dims.y, 0.0f);
    
    if ((i >= dims.x) || (j >= dims.y)) return;
    
    r32 acc = 0.0f;
    r32 amp = 1.0f;
    r32 range = 0.0f;
    for (i32 k = 0; k < n; ++k)
    {
        // 2D noise for now...
        acc += SamplePerlinNoise(pos.x * scale, pos.y * scale, pos.z * scale, 1.0f, ((k + 38) * 27389482)) * amp;
        
        scale *= lacunarity;
        range += PERLIN_3D_RANGE * amp;
        amp *= decay;
    }
    
    fb[pixel_index] = inv_lerp(-range, range, acc);
}

GLOBAL
void PerlinTurbulenceKernel(r32 *fb, float3 dims, i32 seed, i32 n, r32 scale, r32 lacunarity, r32 decay)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;
    int pixel_index = j * dims.x + i;
    
    float3 pos = make_float3((r32)i / (r32)dims.x, (r32)j / (r32)dims.y, 0.0f);
    
    if ((i >= dims.x) || (j >= dims.y)) return;
    
    r32 acc = 0.0f;
    r32 amp = 1.0f;
    r32 range = PERLIN_3D_RANGE;
    for (i32 k = 0; k < n; ++k)
    {
        // 2D noise for now...
        //acc += SamplePerlinNoise(pos.x * scale, pos.y * scale, pos.z * scale, 1.0f, ((k + 38) * 27389482)) * amp;
        acc += fabsf(SamplePerlinNoise(pos.x * scale, pos.y * scale, pos.z * scale, 1.0f, seed)) * amp;
        scale *= lacunarity;
        amp *= decay;
        range += PERLIN_3D_RANGE * amp;
    }
    
    // TODO(Dustin): I am not convinced this is working...
    fb[pixel_index] = inv_lerp(0.0f, range, acc);
}

//Predeclare these functions...TEMPORARY
void mprint(char *fmt, ...);
void mprinte(char *fmt, ...);

#include <Platform/Timer.h>

void PerlinKernelEntry(uchar4 *pbo, i32 texture_width, i32 texture_height)
{
    int nx = texture_width;
    int ny = texture_height;
    int tx = 16;
    int ty = 16;
    int num_pixels = nx * ny;
    size_t fb_size = sizeof(r32) * num_pixels;
    dim3 blocks(nx/tx+1, ny/ty+1);
    dim3 threads(tx,ty);
    
    // Image to write to
    r32* cudaimage = NULL;
    CheckCudaErrors(hipMalloc((void**)&cudaimage, fb_size));
    
    Timer timer;
    timer.Begin();
    
    PerlinKernel<<<blocks, threads>>>(cudaimage, 
                                      make_float3(nx, ny, 0), 
                                      0x71889283, 
                                      8,
                                      5.9f,  // "Zoom" 
                                      2.0f,    // "Noisy"
                                      0.5f);
    
    CheckCudaErrors(hipGetLastError());
    CheckCudaErrors(hipDeviceSynchronize());
    
    r32 elapsed = timer.MiliSecondsElapsed();
    mprint("Ms elapsed: %lf\n", elapsed);
    
    SendImageToPboR<<<blocks, threads>>>(pbo, nx, ny, cudaimage);
    CheckCudaErrors(hipGetLastError());
    CheckCudaErrors(hipDeviceSynchronize());
    
    //retrieve image from GPU
    r32 *image = new r32[num_pixels];
    CheckCudaErrors(hipMemcpy(image, cudaimage, nx * ny * sizeof(r32), hipMemcpyDeviceToHost));
    
    CheckCudaErrors(hipFree(cudaimage));
    hipDeviceReset();
}